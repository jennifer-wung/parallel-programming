#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define INF 	1000000
#define CMCPYHTD hipMemcpyHostToDevice
#define CMCPYDTH hipMemcpyDeviceToHost

#define THREAD_WIDTH 2
#define BLOCK_WIDTH 16

#define HANDLE_ERROR(status) \
{ \
	if (status != hipSuccess) \
	{ \
		fprintf(stderr, "%s failed  at line %d \nError message: %s \n", \
			__FILE__, __LINE__ ,hipGetErrorString(status)); \
		exit(EXIT_FAILURE); \
	} \
}

const int V = 7000;
bool gDebug = false;
int n, m;       // Number of vertices, edges
int Dist[V*V];


template <int BLOCK_FACTOR> __global__ void cal_phase_one(const unsigned int Round, const unsigned int n, const size_t pitch, int * const d) 
{
	int newPath;
	
	const int tx = threadIdx.x; 
	const int ty = threadIdx.y;

	// To calculate original index of elements in the block, 0 <= ty, tx < BLOCK_FACTOR
	const int v1 = BLOCK_FACTOR*Round + ty; // block_internal_y 
	const int v2 = BLOCK_FACTOR*Round + tx; // block_internal_x

	const int weightID = v1*pitch + v2;

	__shared__ int primary_d[BLOCK_FACTOR][BLOCK_FACTOR];
	//int primary_d[BLOCK_FACTOR][BLOCK_FACTOR];

	if (v1 < n && v2 < n) {
		primary_d[ty][tx] = d[weightID];
	} else {
		primary_d[ty][tx] = INF;
	}

	// Synchronize to make sure the all value are loaded in block
	__syncthreads();
	
	// For each block, it need to compute B times
	#pragma unroll
	for (int i = 0; i < BLOCK_FACTOR; i++) {
		newPath = primary_d[ty][i] + primary_d[i][tx];
		__syncthreads();
		if (newPath < primary_d[ty][tx]) {
			primary_d[ty][tx] = newPath;
		}
		// Synchronize to make sure that all value are current
		__syncthreads();
	}
	
	if (v1 < n && v2 < n) {
		d[weightID] = primary_d[ty][tx];
	}
}

template <int BLOCK_FACTOR> __global__ void cal_phase_two(const unsigned int Round, const unsigned int n, const size_t pitch, int * const d)
{
	// done calculation in phase 1
	if(blockIdx.x == Round) return;		
	int newPath;

        const int tx = threadIdx.x;
        const int ty = threadIdx.y;

        // To calculate original index of elements in the block, 0 <= ty, tx < BLOCK_FACTOR
	// so each cuda block will have its own pivot-block results calculated from phase 1
        int v1 = BLOCK_FACTOR*Round + ty; // block_internal_y
        int v2 = BLOCK_FACTOR*Round + tx; // block_internal_x

	// Shared varialbes are shared within each B*B submatrice
	__shared__ int primary_d[BLOCK_FACTOR][BLOCK_FACTOR];
	__shared__ int current_d[BLOCK_FACTOR][BLOCK_FACTOR];	
	

	// Pivot-block (The result of pivot-row (pivot-column) blocks depends on pivot block in phase 1 and itself)	
	const int cell_primary = v1 * pitch + v2;
	if (v1 < n && v2 < n) {
		primary_d[ty][tx] = d[cell_primary];
	} else {
                primary_d[ty][tx] = INF;
	}
	
	// Pivot-row and pivot-column blocks
	if (blockIdx.y == 0) { // pivot-row blcoks
		v1 = BLOCK_FACTOR*Round + ty;
		v2 = BLOCK_FACTOR*blockIdx.x + tx;
	} else { // pivot-col blocks
		v1 = BLOCK_FACTOR*blockIdx.x + ty;
		v2 = BLOCK_FACTOR*Round + tx;
	}		

	const int cell_current = v1*pitch + v2;
	if (v1 < n && v2 < n) {
		current_d[ty][tx] = d[cell_current];
	} else {
		current_d[ty][tx] = INF;
	} 
	// Synchronize to make sure the all value are loaded in block
	__syncthreads();

	if (blockIdx.y == 0) { // pivot-row blcoks
		#pragma unroll
		for (int i = 0; i < BLOCK_FACTOR; i++) {
			newPath = primary_d[ty][i] + current_d[i][tx];
			__syncthreads();
			if (newPath < current_d[ty][tx]) {
				current_d[ty][tx] = newPath;
			}
			__syncthreads();
		}

	} else { // pivot-col blocks
		#pragma unroll
		for(int i = 0; i < BLOCK_FACTOR; i++) {
			newPath = current_d[ty][i] + primary_d[i][tx];
			__syncthreads();
                        if (newPath < current_d[ty][tx]) {
                                current_d[ty][tx] = newPath;
                        }
			__syncthreads();
		}
	}

	if (v1 < n && v2 < n) {
		d[cell_current] = current_d[ty][tx];
	}
}

template <int BLOCK_SIZE, int THREAD_SIZE> __global__ void cal_phase_three(const unsigned int Round, const unsigned int n, const size_t pitch, int * const d) 
{
	if(blockIdx.x == Round || blockIdx.y == Round) return;
	int newPath;
	int path;

	const int tx = threadIdx.x*THREAD_SIZE;
	const int ty = threadIdx.y*THREAD_SIZE;

	//the ID of the rest of the blocks (no blocks that have been calculated in phase 1 & 2)
  	const int v1 = blockDim.y*blockIdx.y*THREAD_SIZE + ty;
	const int v2 = blockDim.x*blockIdx.x*THREAD_SIZE + tx;
	
	__shared__ int primaryRow_d[BLOCK_SIZE * THREAD_SIZE][BLOCK_SIZE * THREAD_SIZE];
	__shared__ int primaryCol_d[BLOCK_SIZE * THREAD_SIZE][BLOCK_SIZE * THREAD_SIZE];
	
	// pivot-row and pivot-col from phase 1 & 2
	int v1Row = BLOCK_SIZE*Round*THREAD_SIZE + ty; 
	int v2Col = BLOCK_SIZE*Round*THREAD_SIZE + tx;
	
	int idx, idy;
 	int weightID;
	// each cuda block has its own pivot-block results calculated from phase 1 and phase 2 (like a cross shape)
	#pragma unroll
	for (int i = 0; i < THREAD_SIZE; i++) {
		#pragma unroll
		for (int j = 0; j < THREAD_SIZE; j++) {
			idx = tx + j;
			idy = ty + i;
			if(v1Row+i < n && v2+j < n) {
				weightID = (v1Row+i)*pitch + v2 + j;
				primaryRow_d[idy][idx] = d[weightID];
			} else {
				primaryRow_d[idy][idx] = INF;
			}

			if (v1 + i  < n && v2Col + j < n)
			{
				weightID = (v1 + i) * pitch + v2Col + j;
				primaryCol_d[idy][idx] = d[weightID];
			}
			else
			{
				primaryCol_d[idy][idx] = INF;
			}
		}
	}
	__syncthreads();
	#pragma unroll
        for (int i = 0; i < THREAD_SIZE; i++) {
                #pragma unroll
                for (int j = 0; j < THREAD_SIZE; j++) { 
			if(v1+i < n && v2 +j < n) {
				weightID = (v1+i)*pitch + v2 + j;
				path = d[weightID];
				idx = tx + j;
                        	idy = ty + i;
				
				#pragma unroll
				for(int k=0; k < BLOCK_SIZE*THREAD_SIZE; k++) {
					newPath = primaryCol_d[idy][k] + primaryRow_d[k][idx];
					if(path > newPath) {
						path = newPath;
						
					}
					d[weightID] = path;
				}		
				//d[weightID] = path;
			}
		}
	}
}	


int ceil(int a, int b)
{       return (a + b -1)/b;
}

template <int BLOCK_SIZE, int THREAD_SIZE> void block_APSP()
{
	int *dev_d = 0;

	size_t pitch;
	size_t pitch_int;

	// Size of blocking_factor
	const int BLOCK_FACTOR = BLOCK_SIZE * THREAD_SIZE;
	int round = ceil(n, BLOCK_FACTOR);
	printf("FW-Round = %d\n",round);

	hipError_t cudaStatus;
	hipStream_t cpyStream;

	// Initialize the grid and block dimensions here
	dim3 dimGridP1(1, 1, 1);
	dim3 dimGridP2(round, 2, 1);
	dim3 dimGridP3(round, round, 1);
	
	dim3 dimBlockP1(BLOCK_FACTOR, BLOCK_FACTOR, 1);
	dim3 dimBlockP2(BLOCK_FACTOR, BLOCK_FACTOR, 1);
	dim3 dimBlockP3(BLOCK_SIZE, BLOCK_SIZE, 1);
	
	if (gDebug)
        {
                printf("|V| %d\n", n);

                printf("Phase 1\n");
                printf("Dim Grid:\nx - %d\ny - %d\nz - %d\n", dimGridP1.x, dimGridP1.y, dimGridP1.z);
                printf("Dim Block::\nx - %d\ny - %d\nz - %d\n", dimBlockP1.x, dimBlockP1.y, dimBlockP1.z);

                printf("\nPhase 2\n");
                printf("Dim Grid:\nx - %d\ny - %d\nz - %d\n", dimGridP2.x, dimGridP2.y, dimGridP2.z);
                printf("Dim Block::\nx - %d\ny - %d\nz - %d\n", dimBlockP2.x, dimBlockP2.y, dimBlockP2.z);

                printf("Phase 3\n");
                printf("Dim Grid:\nx - %d\ny - %d\nz - %d\n", dimGridP3.x, dimGridP3.y, dimGridP3.z);
                printf("Dim Block::\nx - %d\ny - %d\nz - %d\n", dimBlockP3.x, dimBlockP3.y, dimBlockP3.z);
        }

	// Create new stream to copy data	
	cudaStatus = hipStreamCreate(&cpyStream);
	HANDLE_ERROR(cudaStatus);
	// Allocate GPU buffers for matrix of shortest paths d(G)
	cudaStatus = hipMallocPitch(&dev_d, &pitch, n*sizeof(int), n);
	HANDLE_ERROR(cudaStatus);	
		
	pitch_int = pitch / sizeof(int);
	//printf("pitch_int = %d\n", pitch_int);
	
	hipMemcpy2DAsync(dev_d, pitch, Dist, n*sizeof(int), n*sizeof(int), n, CMCPYHTD, cpyStream);

	for(int r = 0; r < round; r++) { 
		cal_phase_one<BLOCK_FACTOR><<<1, dimBlockP1>>>(r, n, pitch_int, dev_d);
		cal_phase_two<BLOCK_FACTOR><<<dimGridP2, dimBlockP2>>>(r, n, pitch_int, dev_d);
		cal_phase_three<BLOCK_SIZE, THREAD_SIZE><<<dimGridP3, dimBlockP3>>>(r, n, pitch_int, dev_d);	
	}

	// Check for any errors launching the kernel
    	cudaStatus = hipGetLastError();
	HANDLE_ERROR(cudaStatus);

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
    	// any errors encountered during the launch.
    	cudaStatus = hipDeviceSynchronize();
	HANDLE_ERROR(cudaStatus);

	cudaStatus = hipMemcpy2D(Dist, n*sizeof(int), dev_d, pitch, n*sizeof(int), n, CMCPYDTH);
	HANDLE_ERROR(cudaStatus);
	
	cudaStatus = hipFree(dev_d);
	HANDLE_ERROR(cudaStatus);
	
	return;
}

void print_graph()
{
	for(int v1 = 0; v1 < n; v1++)
	{
		for (int v2 = 0; v2 < n; v2++ )
		{	
			if (Dist[v1 * n + v2] < INF)
				printf("%d ", Dist[v1 * n + v2]);
			else
				printf("INF ");
		}
		printf("\n");
	}
	printf("\n");
}

void input(char *inFileName)
{
        FILE *infile = fopen(inFileName, "r");
        fscanf(infile, "%d %d", &n, &m);

        for (int i = 0; i < n*(n+1); ++i)
                Dist[i]=INF;

        for (int i = 0; i < n; ++i)
                Dist[i*n + i ] = 0;
        while (--m >= 0) {
                int v1, v2, w;
                fscanf(infile, "%d %d %d", &v1, &v2, &w);
                Dist[(v1-1) * n + (v2-1)] = w;
        }
}

void output(char *outFileName)
{       FILE *outfile = fopen(outFileName, "w");
        for (int i = 0; i < n; ++i) {
                for (int j = 0; j < n; ++j) {
                        if (Dist[i*n + j] >= INF)  fprintf(outfile, "INF ");
                        else fprintf(outfile, "%d ", Dist[i*n+j]);
                }
                fprintf(outfile, "\n");
        }
}


int init_device()
{	// Task 1: Device Initialization
	int deviceCount;
	//int i;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("\n\tNo CUDA device found.\n\n");
		return 0;
	} else {	
		for( int i = 0; i < deviceCount; i++) {
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, i);
			//printf("sharedMemPerBlock: %d\n", prop.sharedMemPerBlock);
			//printf("maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
		}
		//printf("deviceCount: %d\n",deviceCount);
		hipSetDevice(0);
    		return 0;
	}
}

int main(int argc, char *argv[]) 
{

	init_device();
	input(argv[1]);
	
	// Initialize CUDA Event
	hipEvent_t start,stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	block_APSP<BLOCK_WIDTH, THREAD_WIDTH>();

	// Finish recording
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	// Calculate elasped time
	hipEventElapsedTime(&elapsedTime,start,stop);
	elapsedTime /= 1000;
	printf ("Time : %f ms\n", elapsedTime*1000);

	output(argv[2]);

	return 0;
}
