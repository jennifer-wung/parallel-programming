#include <stdio.h>
#include <stdlib.h>
#include <omp.h>    /* for OpenMP */
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define INF 	1000000
#define CMCPYHTD hipMemcpyHostToDevice
#define CMCPYDTH hipMemcpyDeviceToHost
#define CMCPYHTH hipMemcpyHostToHost

#define THREAD_WIDTH 2
#define BLOCK_WIDTH 16

#define HANDLE_ERROR(status) \
{ \
	if (status != hipSuccess) \
	{ \
		fprintf(stderr, "%s failed  at line %d \nError message: %s \n", \
			__FILE__, __LINE__ ,hipGetErrorString(status)); \
		exit(EXIT_FAILURE); \
	} \
}

const int V = 7000;
bool gDebug = false;
int n, m;       // Number of vertices, edges
int Dist[V*V];

int* combineData(int numRound, int r, int* d0, int* d1, int p)
{
        int BLOCK_FACTOR = THREAD_WIDTH*BLOCK_WIDTH;
        if (p == 1) { // d0: upper part;  d1: lower half
                for (int i = 0; i < r*BLOCK_FACTOR; i++)
                        d1[i] = d0[i];
        } else {
                for (int i = (r+1)*BLOCK_FACTOR; i < numRound*BLOCK_FACTOR; i++)
                        d0[i] = d1[i];
		//return d0;
        }
	return d1;
}


template <int BLOCK_FACTOR> __global__ void cal_phase_one(const unsigned int Round, const unsigned int n, const size_t pitch, int * const d) 
{
	int newPath;
	
	const int tx = threadIdx.x; 
	const int ty = threadIdx.y;

	// To calculate original index of elements in the block, 0 <= ty, tx < BLOCK_FACTOR
	const int v1 = BLOCK_FACTOR*Round + ty; // block_internal_y 
	const int v2 = BLOCK_FACTOR*Round + tx; // block_internal_x

	const int weightID = v1*pitch + v2;

	__shared__ int primary_d[BLOCK_FACTOR][BLOCK_FACTOR];

	if (v1 < n && v2 < n) {
		primary_d[ty][tx] = d[weightID];
	} else {
		primary_d[ty][tx] = INF;
	}

	// Synchronize to make sure the all value are loaded in block
	__syncthreads();
	
	// For each block, it need to compute B times
	#pragma unroll
	for (int i = 0; i < BLOCK_FACTOR; i++) {
		newPath = primary_d[ty][i] + primary_d[i][tx];
		__syncthreads();
		if (newPath < primary_d[ty][tx]) {
			primary_d[ty][tx] = newPath;
		}
		// Synchronize to make sure that all value are current
		__syncthreads();
	}
	
	if (v1 < n && v2 < n) {
		d[weightID] = primary_d[ty][tx];
	}
}

template <int BLOCK_FACTOR> __global__ void cal_phase_two(const unsigned int Round, const unsigned int n, const size_t pitch, int * const d, const unsigned int id)
{
	// done calculation in phase 1
	if(blockIdx.x == Round) return;		
	int newPath;

        const int tx = threadIdx.x;
        const int ty = threadIdx.y;

        // To calculate original index of elements in the block, 0 <= ty, tx < BLOCK_FACTOR
	// so each cuda block will have its own pivot-block results calculated from phase 1
        int v1 = BLOCK_FACTOR*Round + ty; // block_internal_y
        int v2 = BLOCK_FACTOR*Round + tx; // block_internal_x

	// Shared varialbes are shared within each B*B submatrice
	__shared__ int primary_d[BLOCK_FACTOR][BLOCK_FACTOR];
	__shared__ int current_d[BLOCK_FACTOR][BLOCK_FACTOR];	
	

	// Pivot-block (The result of pivot-row (pivot-column) blocks depends on pivot block in phase 1 and itself)	
	const int cell_primary = v1 * pitch + v2;
	if (v1 < n && v2 < n) {
		primary_d[ty][tx] = d[cell_primary];
	} else {
                primary_d[ty][tx] = INF;
	}
	
	// Pivot-row and pivot-column blocks
	if (blockIdx.y == 0) { // pivot-row blcoks
		v1 = BLOCK_FACTOR*Round + ty;
		v2 = BLOCK_FACTOR*blockIdx.x + tx;
	} else { // pivot-col blocks
		v1 = BLOCK_FACTOR*blockIdx.x + ty;
		v2 = BLOCK_FACTOR*Round + tx;
	}		

	const int cell_current = v1*pitch + v2;
	if (v1 < n && v2 < n) {
		current_d[ty][tx] = d[cell_current];
	} else {
		current_d[ty][tx] = INF;
	} 
	// Synchronize to make sure the all value are loaded in block
	__syncthreads();

	if (blockIdx.y == 0) { // pivot-row blcoks
		#pragma unroll
		for (int i = 0; i < BLOCK_FACTOR; i++) {
			newPath = primary_d[ty][i] + current_d[i][tx];
			__syncthreads();
			if (newPath < current_d[ty][tx]) {
				current_d[ty][tx] = newPath;
			}
			__syncthreads();
		}

	} else { // pivot-col blocks
		#pragma unroll
		for(int i = 0; i < BLOCK_FACTOR; i++) {
			newPath = current_d[ty][i] + primary_d[i][tx];
			__syncthreads();
                        if (newPath < current_d[ty][tx]) {
                                current_d[ty][tx] = newPath;
                        }
			__syncthreads();
		}
	}

	if (v1 < n && v2 < n) {
		d[cell_current] = current_d[ty][tx];
	}
}

template <int BLOCK_SIZE, int THREAD_SIZE> __global__ void cal_phase_three(const unsigned int Round, const unsigned int n, const size_t pitch, int * const d, const unsigned int id) 
{
	if(blockIdx.x == Round || blockIdx.y == Round) return;
	if(id == 0)
		if(blockIdx.y > Round) return;
	else
		if(blockIdx.y < Round) return;

	int newPath;
	int path;

	const int tx = threadIdx.x*THREAD_SIZE;
	const int ty = threadIdx.y*THREAD_SIZE;

	//the ID of the rest of the blocks (no blocks that have been calculated in phase 1 & 2)
  	const int v1 = blockDim.y*blockIdx.y*THREAD_SIZE + ty;
	const int v2 = blockDim.x*blockIdx.x*THREAD_SIZE + tx;
	
	__shared__ int primaryRow_d[BLOCK_SIZE * THREAD_SIZE][BLOCK_SIZE * THREAD_SIZE];
	__shared__ int primaryCol_d[BLOCK_SIZE * THREAD_SIZE][BLOCK_SIZE * THREAD_SIZE];
	
	// pivot-row and pivot-col from phase 1 & 2
	int v1Row = BLOCK_SIZE*Round*THREAD_SIZE + ty; 
	int v2Col = BLOCK_SIZE*Round*THREAD_SIZE + tx;
	
	int idx, idy;
 	int weightID;
	// each cuda block has its own pivot-block results calculated from phase 1 and phase 2 (like a cross shape)
	#pragma unroll
	for (int i = 0; i < THREAD_SIZE; i++) {
		#pragma unroll
		for (int j = 0; j < THREAD_SIZE; j++) {
			idx = tx + j;
			idy = ty + i;
			if(v1Row+i < n && v2+j < n) {
				weightID = (v1Row+i)*pitch + v2 + j;
				primaryRow_d[idy][idx] = d[weightID];
			} else {
				primaryRow_d[idy][idx] = INF;
			}

			if (v1 + i  < n && v2Col + j < n)
			{
				weightID = (v1 + i) * pitch + v2Col + j;
				primaryCol_d[idy][idx] = d[weightID];
			}
			else
			{
				primaryCol_d[idy][idx] = INF;
			}
		}
	}
	__syncthreads();
	#pragma unroll
        for (int i = 0; i < THREAD_SIZE; i++) {
                #pragma unroll
                for (int j = 0; j < THREAD_SIZE; j++) { 
			if(v1+i < n && v2 +j < n) {
				weightID = (v1+i)*pitch + v2 + j;
				path = d[weightID];
				idx = tx + j;
                        	idy = ty + i;
				
				#pragma unroll
				for(int k=0; k < BLOCK_SIZE*THREAD_SIZE; k++) {
					newPath = primaryCol_d[idy][k] + primaryRow_d[k][idx];
					if(path > newPath) {
						path = newPath;
						
					}
					d[weightID] = path;
				}		
				//d[weightID] = path;
			}
		}
	}
}	


int ceil(int a, int b)
{       return (a + b -1)/b;
}

template <int BLOCK_SIZE, int THREAD_SIZE> void block_APSP()
{
	int num_gpus;
        hipGetDeviceCount(&num_gpus);
	omp_set_num_threads(num_gpus);

	size_t pitch;
	size_t pitch_int;

	// Size of blocking_factor
	const int BLOCK_FACTOR = BLOCK_SIZE * THREAD_SIZE;
	int round = ceil(n, BLOCK_FACTOR);
	printf("FW-Round = %d\n",round);

	hipError_t cudaStatus;
	hipStream_t cpyStream;
	//cudaStream_t stream[num_gpus];

	// Initialize the grid and block dimensions here
	dim3 dimGridP1(1, 1, 1);
	dim3 dimGridP2(round, 2, 1);
	dim3 dimGridP3(round, round, 1);
	
	dim3 dimBlockP1(BLOCK_FACTOR, BLOCK_FACTOR, 1);
	dim3 dimBlockP2(BLOCK_FACTOR, BLOCK_FACTOR, 1);
	dim3 dimBlockP3(BLOCK_SIZE, BLOCK_SIZE, 1);
	
	if (gDebug)
        {
                printf("|V| %d\n", n);

                printf("Phase 1\n");
                printf("Dim Grid:\nx - %d\ny - %d\nz - %d\n", dimGridP1.x, dimGridP1.y, dimGridP1.z);
                printf("Dim Block::\nx - %d\ny - %d\nz - %d\n", dimBlockP1.x, dimBlockP1.y, dimBlockP1.z);

                printf("\nPhase 2\n");
                printf("Dim Grid:\nx - %d\ny - %d\nz - %d\n", dimGridP2.x, dimGridP2.y, dimGridP2.z);
                printf("Dim Block::\nx - %d\ny - %d\nz - %d\n", dimBlockP2.x, dimBlockP2.y, dimBlockP2.z);

                printf("Phase 3\n");
                printf("Dim Grid:\nx - %d\ny - %d\nz - %d\n", dimGridP3.x, dimGridP3.y, dimGridP3.z);
                printf("Dim Block::\nx - %d\ny - %d\nz - %d\n", dimBlockP3.x, dimBlockP3.y, dimBlockP3.z);
        }

	// Create new stream to copy data	
	cudaStatus = hipStreamCreate(&cpyStream);
	HANDLE_ERROR(cudaStatus);
	
	int *host_d0 = 0;
	int *host_d1 = 0;
        hipHostMalloc( (void **) &host_d0, n*n*sizeof(int), hipHostMallocDefault, hipHostMallocDefault);
	hipHostMalloc( (void **) &host_d1, n*n*sizeof(int), hipHostMallocDefault, hipHostMallocDefault);
    #pragma omp parallel 
    {
  	int p = omp_get_thread_num();
  	hipSetDevice(p);
	//int gpu_id = -1;
        //cudaGetDevice(&gpu_id);		
	//printf("p = %d\n",gpu_id);
    	int *dev_d = 0;
	// Allocate GPU buffers for matrix of shortest paths d(G)
	cudaStatus = hipMallocPitch(&dev_d, &pitch, n*sizeof(int), n);
	HANDLE_ERROR(cudaStatus);	
		
	pitch_int = pitch / sizeof(int);
	hipMemcpy2D(dev_d, pitch, Dist, n*sizeof(int), n*sizeof(int), n, CMCPYHTD);
	cudaStatus = hipMemcpy2D(host_d0, n*sizeof(int), dev_d, pitch, n*sizeof(int), n, CMCPYDTH);
	cudaStatus = hipMemcpy2D(host_d1, n*sizeof(int), dev_d, pitch, n*sizeof(int), n, CMCPYDTH);
	
	for(int r = 0; r < round; r++) { 
		cal_phase_one<BLOCK_FACTOR><<<1, dimBlockP1>>>(r, n, pitch_int, dev_d);
		cal_phase_two<BLOCK_FACTOR><<<dimGridP2, dimBlockP2>>>(r, n, pitch_int, dev_d, p);
  		hipSetDevice(p);
		cal_phase_three<BLOCK_SIZE, THREAD_SIZE><<<dimGridP3, dimBlockP3>>>(r, n, pitch_int, dev_d, p);
		//cal_phase_three<BLOCK_SIZE, THREAD_SIZE><<<dimGridP3, dimBlockP3>>>(r, n, pitch_int, dev_d, 1);
	   cudaStatus = hipDeviceSynchronize();
	   #pragma omp barrier 
	   {
		//if (p==0) {// did the upper part
		//	cudaStatus = cudaMemcpy2D(host_d0, n*sizeof(int), dev_d, pitch, n*sizeof(int), n, CMCPYDTH);
			//cudaStatus = cudaDeviceSynchronize();
                        //for (int i = (r+1)*BLOCK_FACTOR; i < round*BLOCK_FACTOR; i++)
                        //	host_d0[i] = host_d1[i];
		//	HANDLE_ERROR(cudaStatus);
		//} else {
		if(p==1) {
			cudaStatus = hipMemcpy2D(host_d1, n*sizeof(int), dev_d, pitch, n*sizeof(int), n, CMCPYDTH);
			//cudaStatus = cudaDeviceSynchronize();
                        //for (int i = 0; i < r*BLOCK_FACTOR; i++)
                        //	host_d1[i] = host_d0[i];
			
			HANDLE_ERROR(cudaStatus);
		}
		cudaStatus = hipDeviceSynchronize();
		HANDLE_ERROR(cudaStatus);
		#pragma omp barrier
		{
                        cudaStatus = hipMemcpy2D(dev_d, pitch, host_d1, n*sizeof(int), n*sizeof(int), n, CMCPYHTD);
		}
		HANDLE_ERROR(cudaStatus);
	   }
	}

	// Check for any errors launching the kernel
    	cudaStatus = hipGetLastError();
	HANDLE_ERROR(cudaStatus);

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
    	// any errors encountered during the launch.
 	cudaStatus = hipDeviceSynchronize();
	HANDLE_ERROR(cudaStatus);
   	#pragma omp master 
		cudaStatus = hipMemcpy2D(Dist, n*sizeof(int), dev_d, pitch, n*sizeof(int), n, CMCPYDTH);
	HANDLE_ERROR(cudaStatus);
	cudaStatus = hipFree(dev_d);
  	HANDLE_ERROR(cudaStatus);
	
   }
	return;
}

void print_graph()
{
	for(int v1 = 0; v1 < n; v1++)
	{
		for (int v2 = 0; v2 < n; v2++ )
		{	
			if (Dist[v1 * n + v2] < INF)
				printf("%d ", Dist[v1 * n + v2]);
			else
				printf("INF ");
		}
		printf("\n");
	}
	printf("\n");
}

void input(char *inFileName)
{
        FILE *infile = fopen(inFileName, "r");
        fscanf(infile, "%d %d", &n, &m);
        //memset(Dist, INF, sizeof(int)*V*V);

        for (int i = 0; i < n*(n+1); ++i)
                Dist[i]=INF;

        for (int i = 0; i < n; ++i)
                Dist[i*n + i ] = 0;
        while (--m >= 0) {
                int v1, v2, w;
                fscanf(infile, "%d %d %d", &v1, &v2, &w);
                Dist[(v1-1) * n + (v2-1)] = w;
        }
}

void output(char *outFileName)
{       FILE *outfile = fopen(outFileName, "w");
        for (int i = 0; i < n; ++i) {
                for (int j = 0; j < n; ++j) {
                        if (Dist[i*n + j] >= INF)  fprintf(outfile, "INF ");
                        else fprintf(outfile, "%d ", Dist[i*n+j]);
                }
                fprintf(outfile, "\n");
        }
}

int main(int argc, char *argv[]) 
{

	input(argv[1]);
	
	// Initialize CUDA Event
	hipEvent_t start,stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	block_APSP<BLOCK_WIDTH, THREAD_WIDTH>();

	// Finish recording
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	// Calculate elasped time
	hipEventElapsedTime(&elapsedTime,start,stop);
	elapsedTime /= 1000;
	printf ("Time : %f ms\n", elapsedTime*1000);

	output(argv[2]);

	return 0;
}
